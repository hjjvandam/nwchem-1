#define BLOCK_DIM 16
#include <stdio.h>
#include <hipblas.h>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <omp.h>
#define type double


__global__ void transpose(type *odata, type *idata, int width, int height)
{
	__shared__ type block[BLOCK_DIM][BLOCK_DIM+1];

	// read the matrix tile into shared memory

	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
	if((xIndex < width) && (yIndex < height))
	{
		unsigned int index_in = yIndex * width + xIndex;
		block[threadIdx.y][threadIdx.x] = idata[index_in];
	}
	__syncthreads();

	// write the transposed matrix tile to global memory
	xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;

	if((xIndex < height) && (yIndex < width))
	{
		unsigned int index_out = yIndex * height + xIndex;
		odata[index_out] = block[threadIdx.x][threadIdx.y];
	}

}
extern "C" void matrix_transpose(type *odata, type *idata, int width, int height)
{
#ifdef printd
	printf("In matrix transpose..");
#endif

	int ndim = 2;
	int lda[2];
	lda[0] = width, lda[1] = height;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasOperation_t aflag = HIPBLAS_OP_T, bflag = HIPBLAS_OP_T;
	double alpha = 1, beta = 0;
	//cublasDgeam(handle,aflag, bflag,width, height, &alpha, idata, width, &beta, idata, width, odata, height);
#ifdef NOHTIME
#include "includes/nohtimestart.h"
#endif

	hipblasDgeam(handle,aflag, bflag,height, width, &alpha, idata, width, &beta, idata, width, odata, height);
#ifdef NOHTIME
#include "includes/nohtimestop.h"
#endif

	return;

}
